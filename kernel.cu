#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Macro para verificar erros CUDA
#define cudaCheckError() {                                           \
    hipError_t e = hipGetLastError();                              \
    if (e != hipSuccess) {                                          \
        printf("CUDA error: %s:%d: '%s'\n", __FILE__, __LINE__,      \
               hipGetErrorString(e));                               \
        exit(EXIT_FAILURE);                                          \
    }                                                                \
}

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main()
{
    hipDeviceReset();

    float *d_a, *d_b, *d_c;
    float *a, *b, *c;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 4096;
    int size = n * sizeof(float);

    // Alocação de memória para os vetores no host
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    // Inicialização dos vetores a e b
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Alocação de memória para os vetores no dispositivo
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    cudaCheckError();

    // Cópia dos vetores a e b do host para o dispositivo
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    cudaCheckError();

    // Início do evento
    hipEventRecord(start, 0);

    // Definição do grid e dos blocos
    int threadsPerBlock = 1024;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Execução do kernel
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();
    cudaCheckError();

    // Fim do evento
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Cálculo do tempo decorrido
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time: %3.3f ms\n", elapsedTime);

    // Cópia do vetor c do dispositivo para o host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    cudaCheckError();

    // Verificação dos resultados
    for (int i = 0; i < n; i++) {
        if (c[i] != a[i] + b[i]) {
            printf("Erro no resultado! c[%d] = %f\n", i, c[i]);
            break;
        }
    }

    // Destruir os eventos
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Liberar memória alocada no dispositivo
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Liberar memória alocada no host
    free(a);
    free(b);
    free(c);

    return 0;
}
