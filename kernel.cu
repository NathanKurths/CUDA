#include "hip/hip_runtime.h"
#include <stdio.h>

int main()
{
    hipDeviceReset();
    int *d_a, *d_b, *d_c;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 4096;
    int size = n * sizeof(int);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    for (int i =0; i < n; i++)
        a[i] = i, b[i] = i;

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    vecAdd <<< ceil (n/1024), 1024  >>> (n, d_a, d_b, d_c);
    hipDeviceSynchronize();
}