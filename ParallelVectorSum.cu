
#include <hip/hip_runtime.h>
#include <stdio.h>

int *a, *b, *c; // host data

__global__ void vecAdd(int *a, int *b, int *c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    hipDeviceReset();
    int *d_a, *d_b, *d_c;
    int n = 256;
    int size = n * sizeof(int);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);


    //malloc da gpu
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    for (int i = 0; i<n; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    vecAdd <<<1,n>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyHostToHost);

    printf("\n Resultado da soma: \n");
    for(int i=0; i < n; i++)
    {
        printf(" %d,", c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}