
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel CUDA
__global__ void copy_vector(float *data, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int aux = 0;

    // Define memória compartilhada com tamanho do bloco
    __shared__ float temp_data[256];

    // Garantir que o índice não saia dos limites
    if (index < N) {
        temp_data[threadIdx.x] = data[index];

        __syncthreads();

        // Calcula a soma parcial para o bloco
        for (int i = 0; i < blockDim.x; i++) {
            aux += temp_data[i];
        }

        // Escreve o resultado de volta na memória global
        data[index] = aux;
    }
}

int main() {
    int N = 1024;  // Tamanho do vetor
    size_t size = N * sizeof(float);

    // Aloca memória no host
    float *h_data = (float *)malloc(size);
    for (int i = 0; i < N; i++) {
        h_data[i] = 1.0f;  // Inicializa com 1 para teste
    }

    // Aloca memória no device
    float *d_data;
    hipMalloc(&d_data, size);

    // Copia dados do host para o device
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    // Define o número de blocos e threads por bloco
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Executa o kernel
    copy_vector<<<blocksPerGrid, threadsPerBlock>>>(d_data, N);

    // Copia o resultado de volta para o host
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

    // Verifica o resultado
    for (int i = 0; i < N; i++) {
        printf("%f ", h_data[i]);
    }
    printf("\n");

    hipFree(d_data);
    free(h_data);

    return 0;
}
