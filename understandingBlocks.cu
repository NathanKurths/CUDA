
#include <hip/hip_runtime.h>
#include<stdio.h>

int *a, *b, *c;

__global__ void block()
{
    int i = blockIdx.x;
    printf("Hi I am the block %d\n", i);
}

int main()
{
    hipDeviceReset();

    block<<<13,1>>>();

    hipDeviceSynchronize();

    return 0;
}