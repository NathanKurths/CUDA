
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void meuKernel (int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main(){
    hipDeviceReset();
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    meuKernel<<<1,1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n", a, b, c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}